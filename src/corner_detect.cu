#include <iostream>
#include <chrono>
#include "cuda_utils.h"
#include "logging.h"
#include "corner_detect.h"
#include <fstream>
#include <string>
#include <iomanip>
#include <cmath>
#include "decode_utils.h"

// test odasf

static Logger gLogger;

using namespace nvinfer1;
using namespace std;

// 加载模型，分配显存和内存
CornerDetect::CornerDetect(const std::string & engine_name,string& bbox)
{
    // select device
    hipSetDevice(device);

    // load TRT-ENGINE
    std::ifstream file(engine_name, std::ios::binary);
    assert(file.good() == true);
    char *trtModelStream = nullptr;
    size_t size = 0;
    file.seekg(0, file.end);
    size = file.tellg();
    file.seekg(0, file.beg);
    trtModelStream = new char[size];
    assert(trtModelStream);
    file.read(trtModelStream, size);
    file.close();

    // build trt context
    runtime = createInferRuntime(gLogger);
    assert(runtime != nullptr);
    engine = runtime->deserializeCudaEngine(trtModelStream, size);
    assert(engine != nullptr);
    context = engine->createExecutionContext();
    assert(context != nullptr);
    delete[] trtModelStream;
    assert(engine->getNbBindings() == 2);

    // set input output idx
    int inputIndex, outputIndex;
    for (int bi = 0; bi < engine->getNbBindings(); bi++)
    {
        if (engine->bindingIsInput(bi) == true)
        {
            inputIndex = bi;
        }
        else
        {
            outputIndex = bi;
        }
    }
    assert(inputIndex == 0);
    assert(outputIndex == 1);
    // Create GPU buffers on device
    CUDA_CHECK(hipMalloc(&buffers[inputIndex], batch_size * input_c * input_w * input_h *sizeof(float)));
    CUDA_CHECK(hipMalloc(&buffers[outputIndex], batch_size * output_size * sizeof(float)));
    // Create stream
    CUDA_CHECK(hipStreamCreate(&stream));
    data = new float[batch_size * input_c * input_w * input_h];
    assert(data != nullptr);
    output_buffer = new float[batch_size * output_size];  
    assert(output_buffer != nullptr);  


    // read bbox_priors
    decode_bbox = new float[7098*4];
    assert(decode_bbox != nullptr);
    std::ifstream in(bbox, std::ios::in | std::ios::binary); 
    in.read((char *)decode_bbox, sizeof(float)*7098*4);
    in.close();


    // malloc related memory 
    output = new float[maxoutobject*13+1];
    HANDLE_ERROR(hipMalloc((void**)&gpu_input,   sizeof(float)*batch_size * output_size));
    HANDLE_ERROR(hipMalloc((void**)&gpu_output, sizeof(float)*(maxoutobject*13+1)));
    HANDLE_ERROR(hipMalloc((void**)&gpu_priors,   sizeof(float)*nums*4));
    HANDLE_ERROR(hipMalloc((void**)&gpu_variances,   sizeof(float)*2));
    HANDLE_ERROR(hipMemcpy(gpu_variances,variances,sizeof(float)*2,hipMemcpyHostToDevice));
}


void CornerDetect::preprocess(string& img_name)
{
    cv::Mat img = cv::imread(img_name);
    cv::Mat out;
    cv::resize(img,out,cv::Size(input_w,input_h));
    int i = 0;
    for (int row = 0; row < input_h; ++row) {
        uchar* uc_pixel = out.data + row * out.step;
        for (int col = 0; col < input_w; ++col) {
            data[0 * 3 * input_h * input_w + i] = (float)uc_pixel[0] - 104.0;
            data[0 * 3 * input_h * input_w + i + input_h * input_w] = (float)uc_pixel[1] -117.0;
            data[0 * 3 * input_h * input_w + i + 2 * input_h * input_w] = (float)uc_pixel[2] -123.0;
            uc_pixel += 3;
            ++i;
        }
    }
    cout << "pre_deal_done" << endl;
    

}


void CornerDetect::postprocess(string& img_name,float conf_thresh,float nms_thresh)
{

   
    
    cv::Mat img = cv::imread(img_name);
    cv::Mat imgraw;
    cv::resize(img,imgraw,cv::Size(416,416));
    int imgh = img.rows;
    int imgw = img.cols;

    float fx = imgw/416.0;
    float fy = imgh/416.0;

    // cuda kernel infer 
    HANDLE_ERROR(hipMemcpy(gpu_input,output_buffer,sizeof(float)*batch_size * output_size,hipMemcpyHostToDevice));  
    HANDLE_ERROR(hipMemcpy(gpu_priors,decode_bbox,sizeof(float)*nums*4,hipMemcpyHostToDevice));
    int threadNum = getThreadNum();
    int blockNum = (nums -0.5)/threadNum +1;
    CalDetection <<<blockNum,threadNum>>>(gpu_input,gpu_output,nums,conf_thres,nums,c,maxoutobject,gpu_priors,gpu_variances);
    HANDLE_ERROR(hipMemcpy(output,gpu_output,sizeof(float)*(maxoutobject*13+1),hipMemcpyDeviceToHost));

    // parse results acording Yolo::Detection format
    std::vector<Yolo::Detection> res;
    std::vector<Yolo::Detection> dst;
    printf("Fill conf condition num is:%d\n",int(output[maxoutobject*13]));

    for (int i=0;i<int(output[maxoutobject*13]);++i)
    {
        Yolo::Detection tmp;
        memcpy(tmp.bbox,&output[i*13+0],sizeof(float)*4);
        memcpy(&tmp.corner,&output[i*13+4],sizeof(float)*8);// 解析时注意顺序
        memcpy(&tmp.conf,&output[i*13+12],sizeof(float)*1);
        
        dst.push_back(tmp);     
    }
    
    // nms
    std::sort(dst.begin(), dst.end(), cmp);
    for (size_t m = 0; m < dst.size(); ++m) {
        auto& item = dst[m];
        res.push_back(item);
        for (size_t n = m + 1; n < dst.size(); ++n) {
            if (iou(item.bbox, dst[n].bbox) > nms_thresh) {
                dst.erase(dst.begin() + n);
                --n;
            }
        }
    }
   
    //  save img with bbox
    for (size_t j = 0; j < 1; j++) {
        cv::Rect r = get_rect(res[j].bbox,fx,fy);
        cv::rectangle(img, r, cv::Scalar(0x27, 0xC1, 0x36), 2);
        cv::putText(img, "lp", cv::Point(r.x, r.y - 1), cv::FONT_HERSHEY_PLAIN, 1.2, cv::Scalar(0xFF, 0xFF, 0xFF), 2);
    }
    cv::imwrite("lp.jpg", img);

    //  save perspective_img
    cv::Mat pers_img;
    cv::Mat out_img;
    pers_img = get_perspective_mat(res[0].corner);
    cv::warpPerspective(imgraw,out_img,pers_img,cv::Size(94,24));
    cv::imwrite("pers_lp.jpg", out_img);
    cout << "save_done" << endl;

}

void CornerDetect::infer()
{
    CUDA_CHECK(hipMemcpyAsync(buffers[0], data,  batch_size * input_c * input_w * input_h * sizeof(float), hipMemcpyHostToDevice, stream));
    context->enqueueV2(buffers, stream, nullptr);
    CUDA_CHECK(hipMemcpyAsync(output_buffer, buffers[1], batch_size * output_size * sizeof(float), hipMemcpyDeviceToHost, stream));
    hipStreamSynchronize(stream);
    cout << "infer_done" << endl;
}

// 释放资源
CornerDetect::~CornerDetect()
{
     
    // Release stream and buffers
    hipStreamDestroy(stream);
    CUDA_CHECK(hipFree(buffers[0]));
    CUDA_CHECK(hipFree(buffers[1]));


    CUDA_CHECK(hipFree(gpu_input));
    CUDA_CHECK(hipFree(gpu_output));
    CUDA_CHECK(hipFree(gpu_priors));
    CUDA_CHECK(hipFree(gpu_variances));
    // Destroy the engine
    context->destroy();
    engine->destroy();
    runtime->destroy();



    delete data;
    delete output_buffer;
    delete decode_bbox;
    delete output;
    
    
}




int main()
{
    string name = "../data/0.jpg"; 
    string bbox_name = "../data/bbox.bin";      
    // CornerDetect pred("../data/CORNER-NEW-MERGE.engine",bbox_name);
    CornerDetect pred("../data/CORNER-TEST.engine",bbox_name);

    for (int i=0;i<1000;++i)
    {
        pred.preprocess(name);  
        pred.infer();
        pred.postprocess(name,0.3,0.5);     
    }
                    
              
    return 0;
}

